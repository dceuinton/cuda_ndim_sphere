#include "hip/hip_runtime.h"
#include "utilities.h"
#include <vector>

using namespace std;

void printTestResults(long nDimensions, double radius, long totalPoints) {
	print("\nResults:");
	print("nDimensions: %ld", nDimensions);
	print("radius: %.2f", radius);
	print("total points: %ld\n", totalPoints);
}

long powerLong(long base, long exponent) {
	long result = 1;
	for (int i = 0; i < exponent; ++i) {
		result *= base;
	}
	return result;
}

void convert(long point, long base, vector<long>& index) {
	const long nDimensions = index.size();
	for (long i = 0; i < nDimensions; ++i) {
		index[i] = 0;
	}

	long id = 0;
	while (point != 0) {
		long remainder = point % base;
		point = point / base;
		index[id] = remainder;
		id++;
	}
}

long countPoints(long nDimensions, double radius) {
	const long halfBase = static_cast<long>(floor(radius));
	const long base = 2 * halfBase + 1;
	const double radiusSquared = radius * radius;
	const long nPointsToTest = powerLong(base, nDimensions);

	debug("countPoints():\nhalfBase: %ld\n, base: %ld\n, radiusSquared: %.2f\n, nPointsToTest: %ld", halfBase, base, radiusSquared, nPointsToTest);

	long count = 0;
	vector<long> index(nDimensions, 0);

	for (long point = 0; point < nPointsToTest; ++point) {
	    convert(point, base, index);
	    double testRadiusSquared = 0;
	    for (long dimension = 0; dimension < nDimensions; ++dimension) {
	        double difference = index[dimension] - halfBase;
	        testRadiusSquared += difference * difference;
	    }
	    if (testRadiusSquared < radiusSquared) {
	    	++count;
	    }
	}

	return count;
}

void runSequentialTestCases(long *dimensions, double* radii) {
	print("Sequential Tests ---------------");
	for (int i = 0; i < 3; i++) {
		long totalPoints = countPoints(dimensions[i], radii[i]);
		printTestResults(dimensions[i], radii[i], totalPoints);
	}
	print("Sequential Tests Over ----------\n");
}

__device__ void convert(long point, long base, long* index, long nDimensions) {
	// Ensure array initialised
	for (int i = 0; i < nDimensions; ++i) {
		index[i] = 0;
	}

	long i = 0;
	while (point != 0) {
		long remainder = point % base;
		point = point / base;
		index[i] = remainder;
		i++;
	}
}

// __device__ int outside(long value, long halfBase, double radiusSquared) {
// 	long difference = value - halfBase;

// 	if (difference * difference < radiusSquared) {
// 		return 0;
// 	} else {
// 		return 1;	
// 	}		
// }

__device__ long getDimensionalValue(long point, long base, long dimension) {
	long result = 0;
	for (int i = 0; i < dimension; i++) {
		result = point % base;
		point = point / base;
	}
	return result; 
} 

__device__ void determineOutside(long id, long dimension, 
								 unsigned long long* pointsLength, double radiusSquared, 
								 int* outsideRecord) {
	if (dimension == 1) {
		if (pointsLength[id] < radiusSquared) {
			outsideRecord[id] = 0;
		} else {
			outsideRecord[id] = 1;
		}
	}
}

__device__ void addComponentToLength(long id, long value, long halfBase, unsigned long long* pointsLength) {
	long difference = value - halfBase;
	long difSq = difference * difference;
	unsigned long long differenceSquared = (unsigned long long)difSq;
	atomicAdd(&pointsLength[id], differenceSquared);
}

__global__ void gpuCountPoints(long nPointsToTest, double radiusSquared, 
							   long halfBase, long base, 
							   unsigned long long* pointsLength, int* outsideRecord) {

	long id = blockIdx.x * blockDim.x + threadIdx.x;
	long dimension = threadIdx.y + 1;

	if (id < nPointsToTest) {
		long dimensionalValue = getDimensionalValue(id, base, dimension);
		addComponentToLength(id, dimensionalValue, halfBase, pointsLength);
		determineOutside(id, dimension, pointsLength, radiusSquared, outsideRecord);
		// atomicAdd(&outsideRecord[id], isOutside);

		// int value = changeValue(dimension);
		// atomicAdd(&outsideRecord[id], dimensionalValue);
	}
}

int main(int argc, char** argv) {
	int nTests = 3;
	long dimensions[] = {1, 2, 3};
	double radii[] = {25.5, 2.05, 1.5};

	runSequentialTestCases(dimensions, radii);

	int testCase = 1;

	if (argc == 2) {
		testCase = atoi(argv[1]);
		print("TestCase is %d", testCase);
	}

	const long halfBase = static_cast<long>(floor(radii[testCase]));
	const long base = 2 * halfBase + 1;
	const long nPointsToTest = powerLong(base, dimensions[testCase]);
	const double radiusSquared = radii[testCase] * radii[testCase];

	debug("gpu settings tc:%d: nPointsToTest: %ld, nDimensions: %ld, radius: %f, base: %ld", 
		testCase, nPointsToTest, dimensions[testCase], radii[testCase], base);

	int nBytesOutsideRecord = sizeof(int) * nPointsToTest;
	int nBytesPointLength = sizeof(unsigned long long) * nPointsToTest;

	unsigned long long* pointsLength = (unsigned long long *)malloc(nBytesPointLength);
	int* outsideRecord = (int *)malloc(nBytesOutsideRecord);
	for (int i = 0; i < nPointsToTest; ++i) {
		outsideRecord[i] = 0;
		pointsLength[i] = 0;
	}

	int* gpuOutsideRecord;
	unsigned long long* gpuPointsLength;

	hipMalloc(&gpuOutsideRecord, nBytesOutsideRecord);
	hipMalloc(&gpuPointsLength, nBytesPointLength);

	hipMemcpy(gpuOutsideRecord, outsideRecord, nBytesOutsideRecord, hipMemcpyHostToDevice);
	hipMemcpy(gpuPointsLength, pointsLength, nBytesPointLength, hipMemcpyHostToDevice);

	int xThreads = 1024/dimensions[testCase];
	int nThreads = xThreads * dimensions[testCase];
	int nBlocks = (nPointsToTest + nThreads - 1) / nThreads;

	dim3 blockDimensions(nThreads, dimensions[testCase], 1);
	dim3 gridDimensions(nBlocks, 1, 1);

	gpuCountPoints<<<gridDimensions, blockDimensions>>>(nPointsToTest, radiusSquared, halfBase, base, gpuPointsLength, gpuOutsideRecord);

	hipMemcpy(outsideRecord, gpuOutsideRecord, nBytesOutsideRecord, hipMemcpyDeviceToHost);

	hipFree(gpuOutsideRecord);

	long count = 0;
	for (int i = 0; i < nPointsToTest; ++i) {
		if (outsideRecord[i] == 0) {
			count++;
		}
	}

	for (int i = 0; i < nPointsToTest; i++) {
		print("%d :: %d", i, outsideRecord[i]);
	}

	free(outsideRecord);

	print("Parallel test %d:", testCase);
	printTestResults(dimensions[testCase], radii[testCase], count);
	
	return 0;
}