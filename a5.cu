#include "hip/hip_runtime.h"
#include "utilities.h"
#include <vector>

using namespace std;

void printTestResults(long nDimensions, double radius, long totalPoints) {
	print("\nResults:");
	print("nDimensions: %ld", nDimensions);
	print("radius: %.2f", radius);
	print("total points: %ld\n", totalPoints);
}

long powerLong(long base, long exponent) {
	long result = 1;
	for (int i = 0; i < exponent; ++i) {
		result *= base;
	}
	return result;
}

void convert(long point, long base, vector<long>& index) {
	const long nDimensions = index.size();
	for (long i = 0; i < nDimensions; ++i) {
		index[i] = 0;
	}

	long id = 0;
	while (point != 0) {
		long remainder = point % base;
		point = point / base;
		index[id] = remainder;
		id++;
	}
}

long countPoints(long nDimensions, double radius) {
	const long halfBase = static_cast<long>(floor(radius));
	const long base = 2 * halfBase + 1;
	const double radiusSquared = radius * radius;
	const long nPointsToTest = powerLong(base, nDimensions);

	debug("countPoints():\nhalfBase: %ld\n, base: %ld\n, radiusSquared: %.2f\n, nPointsToTest: %ld", halfBase, base, radiusSquared, nPointsToTest);

	long count = 0;
	vector<long> index(nDimensions, 0);

	for (long point = 0; point < nPointsToTest; ++point) {
	    convert(point, base, index);
	    double testRadiusSquared = 0;
	    for (long dimension = 0; dimension < nDimensions; ++dimension) {
	        double difference = index[dimension] - halfBase;
	        testRadiusSquared += difference * difference;
	    }
	    if (testRadiusSquared < radiusSquared) {
	    	++count;
	    }
	}

	return count;
}

void runSequentialTestCases(long *dimensions, double* radii) {
	print("Sequential Tests ---------------");
	for (int i = 0; i < 3; i++) {
		long totalPoints = countPoints(dimensions[i], radii[i]);
		printTestResults(dimensions[i], radii[i], totalPoints);
	}
	print("Sequential Tests Over ----------\n");
}

__global__ void gpuCountPoints(long nPointsToTest, long nDimensions, double radius, long base) {

}

int main(int argc, char** argv) {
	int nTests = 3;
	long dimensions[] = {1, 2, 3};
	double radii[] = {25.5, 2.05, 1.5};

	runSequentialTestCases(dimensions, radii);

	int testCase = 0;

	const long halfBase = static_cast<long>(floor(radii[testCase]));
	const long base = 2 * halfBase + 1;
	const long nPointsToTest = powerLong(base, dimensions[testCase]);

	debug("gpu settings tc:%d: nPointsToTest: %ld, nDimensions: %ld, radius: %f, base: %ld", 
		testCase, nPointsToTest, dimensions[testCase], radii[testCase], base);

	
	
	return 0;
}